#include "hip/hip_runtime.h"
//RENESIS by a1

extern "C" {
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
#include "sph/sph_streebog.h"
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_fugue.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "x11/cuda_x11.h"

#define NBN 2

static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

void skein512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *g_hash);
void skein512_cpu_setBlock_80(void *pdata);
void x11_cubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
void x13_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
void streebog_set_target(const uint32_t* ptarget);
void streebog_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t* d_resNonce);

extern "C" void renesis_hash(void *output, const void *input)
{
	unsigned char _ALIGN(128) hash[128] = { 0 };

	sph_gost512_context ctx_gost;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_fugue512_context ctx_fugue;
	sph_simd512_context ctx_simd;

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, input, 80);
	sph_skein512_close(&ctx_skein, (void*)hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, (const void*)hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*)hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, (const void*)hash, 64); 
	sph_simd512_close(&ctx_simd, (void*)hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, (const void*)hash, 64);
	sph_shavite512_close(&ctx_shavite, hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, (const void*)hash, 64); 
	sph_jh512_close(&ctx_jh, (void*)hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, (const void*)hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*)hash);

	sph_fugue512_init(&ctx_fugue);
	sph_fugue512(&ctx_fugue, (const void*)hash, 64);
	sph_fugue512_close(&ctx_fugue, (void*)hash);

	sph_gost512_init(&ctx_gost);
	sph_gost512(&ctx_gost, (const void*)hash, 64);
	sph_gost512_close(&ctx_gost, (void*)hash);

	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_renesis(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done){

	int dev_id = device_map[thr_id];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	uint32_t default_throughput;
	if (device_sm[dev_id] <= 500) default_throughput = 1 << 18;
	else if (device_sm[dev_id] <= 520) default_throughput = 1 << 18;
	else if (device_sm[dev_id]>520) default_throughput = (1 << 19) + (1 << 18);

	uint32_t throughput = cuda_default_throughput(thr_id, default_throughput);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	throughput &= 0xFFFFFF70;

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00ff;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferShared);
		}
		gpulog(LOG_INFO,thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		x11_simd512_cpu_init(thr_id, throughput);


		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 8 * sizeof(uint64_t) * throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)));
		h_resNonce[thr_id] = (uint32_t*) malloc(NBN * sizeof(uint32_t));
		if(h_resNonce[thr_id] == NULL){
			gpulog(LOG_ERR,thr_id,"Host memory allocation failed");
			exit(EXIT_FAILURE);
		}
		sleep(2);
		init[thr_id] = true;
	}

	uint32_t _ALIGN(64) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	skein512_cpu_setBlock_80(endiandata);
	hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t));
	streebog_set_target(ptarget);

	do {
		skein512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
		quark_keccak512_cpu_hash_64(thr_id, throughput, NULL, d_hash[thr_id]);
		x11_simd512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		x11_shavite512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		quark_jh512_cpu_hash_64(thr_id, throughput, NULL, d_hash[thr_id]);		
		x11_cubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		x13_fugue512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		streebog_cpu_hash_64_final(thr_id, throughput, d_hash[thr_id], d_resNonce[thr_id]);
		hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost);

		if (h_resNonce[thr_id][0] != UINT32_MAX){
			const uint32_t Htarg = ptarget[7];
			const uint32_t startNounce = pdata[19];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], startNounce + h_resNonce[thr_id][0]);
			renesis_hash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput + 1;
				work_set_target_ratio(work, vhash64);
				pdata[19] = startNounce + h_resNonce[thr_id][0];
				if (h_resNonce[thr_id][1] != UINT32_MAX) {
					pdata[21] = startNounce+h_resNonce[thr_id][1];
					be32enc(&endiandata[19], pdata[21]);
					renesis_hash(vhash64, endiandata);
					if (bn_hash_target_ratio(vhash64, ptarget) > work->shareratio[0]){
						work_set_target_ratio(work, vhash64);
						xchg(pdata[19],pdata[21]);
					}
					res++;
				}
				return res;
			}
			else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", h_resNonce[thr_id][0]);
				hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t));				
			}
		}

		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > (uint64_t)throughput + pdata[19]));

	*hashes_done = pdata[19] - first_nonce + 1;

	return 0;
}

extern "C" void free_renesis(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	free(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);
	hipFree(d_hash[thr_id]);

	x11_simd512_cpu_free(thr_id);
	hipDeviceSynchronize();
	init[thr_id] = false;
}
